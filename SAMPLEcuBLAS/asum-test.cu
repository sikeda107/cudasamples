#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define N 99

int main(int argc, char **argv){

  hipError_t cudaStat;
  hipblasStatus_t stat;
  int print_limit_width = 20;
  int i;
  int n = N;
  float *device_x;
  float sum_result;
  float host_original[N];
  int incx = 1;

  hipblasHandle_t handle = 0;

  for (i = 0; i < n; i++){
    host_original[i] = 1*3;
  }

  // Create the cuBLAS handle
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }

  // Allocate device memory
  cudaStat = hipMalloc((void **)&device_x, sizeof(float) * n);
  if (cudaStat != hipSuccess) {
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
  }

  // Transfer inputs to the device
  hipblasSetVector(n, sizeof(float), host_original, 1, device_x, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data set failed");
    hipFree (device_x);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  //cublasStatus_t cublasSasum(cublasHandle_t handle, int n, const float *x, int incx, float *result)
  //Read more at: http://docs.nvidia.com/cuda/cublas/index.html#ixzz4wUVIQfLc

  stat = hipblasSasum(handle, n, device_x, incx, &sum_result);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("cublas function failed");
    hipFree (device_x);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  printf("host original:\n");
  for (i = 0; i < n; i++){

    printf("%.0f ", host_original[i]);
    if(i%print_limit_width==(print_limit_width-1))printf("\n");
  }
  printf("\n");

  printf("result:\n%f\n",sum_result);

  printf("...\n");

  hipFree(device_x);
  hipblasDestroy(handle);

  return EXIT_SUCCESS;
}
