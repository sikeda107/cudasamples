#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

int main(int argc, char **argv)
{
    int i;
    float *A, *dA;
    float *X, *dX;
    float *Y, *dY;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;

    // Create the cuBLAS handle
    hipblasCreate(&handle));

    // Allocate device memory
    hipMalloc((void **)&dA, sizeof(float) * M * N);
    hipMalloc((void **)&dX, sizeof(float) * N);
    hipMalloc((void **)&dY, sizeof(float) * M);

    // Transfer inputs to the device
    hipblasSetVector(N, sizeof(float), X, 1, dX, 1);
    hipblasSetVector(M, sizeof(float), Y, 1, dY, 1);
    hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M);

    // Execute the matrix-vector multiplication
    hipblasSasum(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1,
                             &beta, dY, 1));

    // Retrieve the output vector from the device
    hipblasGetVector(M, sizeof(float), dY, 1, Y, 1));

    for (i = 0; i < 10; i++)
    {
        printf("%2.2f\n", Y[i]);
    }

    printf("...\n");

    free(A);
    free(X);
    free(Y);

    hipFree(dA));
    hipFree(dY));
    hipblasDestroy(handle);

    return 0;
}
