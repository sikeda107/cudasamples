#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define N 10000


__global__ void Kernel2(float* device_x,int element_size){
  int i;
  if(threadIdx.x == 1 && blockIdx.x < 1){
    for(i=0;i<element_size;i++){
      device_x[i]++;
      printf("%.0f ",device_x[i]);
    }
  }//#end-if
  else if(threadIdx.x < 1 && blockIdx.x < 1){
    for(i=0;i<element_size;i++);
  }
}
__global__ void Kernel(float* device_x,int element_size){
  int i;
  if(threadIdx.x < 1 && blockIdx.x < 1){
    for(i=0;i<element_size;i++){
      device_x[i]++;
      printf("%.0f ",device_x[i]);
    }
  }//#end-if
  else if(threadIdx.x == 1 && blockIdx.x < 1){
      for(i=0;i<element_size;i++);
  }
}

int main(int argc, char **argv){

  hipError_t cudaStat;
  hipblasStatus_t stat;
  int print_limit_width = 20;
  int i;
  int n = N;
  float *device_x;
  float sum_result;
  float host_original[N];
  int incx = 1;
  int device_number = 0;
  hipDeviceProp_t deviceProp;
  hipblasHandle_t handle = 0;

  //----set up device START-----
  hipGetDeviceProperties(&deviceProp,device_number);
  //  printf("Using Device %d:%s\n",dev_num,deviceProp.name);
  hipSetDevice(device_number);
  //----set up device END-----


  for (i = 0; i < n; i++){
    host_original[i] = 1;
  }

  // Create the cuBLAS handle
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }

  // Allocate device memory
  cudaStat = hipMalloc((void **)&device_x, sizeof(float) * n);
  if (cudaStat != hipSuccess) {
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
  }

  // Transfer inputs to the device
  hipblasSetVector(n, sizeof(float), host_original, 1, device_x, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data set failed");
    hipFree (device_x);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  dim3 block(2);
  dim3 grid(1);
  Kernel<<<grid,block>>>(device_x,n);
  Kernel2<<<grid,block>>>(device_x,n);
  //cublasStatus_t cublasSasum(cublasHandle_t handle, int n, const float *x, int incx, float *result)
  //Read more at: http://docs.nvidia.com/cuda/cublas/index.html#ixzz4wUVIQfLc

  stat = hipblasSasum(handle, n, device_x, incx, &sum_result);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("cublas function failed");
    hipFree (device_x);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  Kernel<<<grid,block>>>(device_x,n);
  Kernel2<<<grid,block>>>(device_x,n);

  stat = hipblasSasum(handle, n, device_x, incx, &sum_result);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("cublas function failed");
    hipFree (device_x);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  printf("host original:\n");
  for (i = 0; i < n; i++){

    printf("%.0f ", host_original[i]);
    if(i%print_limit_width==(print_limit_width-1))printf("\n");
  }
  printf("\n");

  printf("result:\n%f\n",sum_result);

  printf("...\n");

  hipFree(device_x);
  hipblasDestroy(handle);

  return EXIT_SUCCESS;
}
