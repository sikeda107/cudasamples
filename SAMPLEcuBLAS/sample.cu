
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<hipblas.h>
#define N 100
#define M 150
#define K 50
#define IDX2C(i,j,ld) (((j)*(ld)+(i)))
int main(int argc,char **argv){
  double alpha = 3.0, beta = 1.0;
  double *A,*B,*C;
  double *dA,*dB,*dC;
  int LDA = M, LDB = K, LDC = M;
  int i,j;
  hipSetDevice(0);
  cublasInit();
  hipHostMalloc((void **)&A,sizeof(double) * M * K, hipHostMallocDefault);
  hipHostMalloc((void **)&B,sizeof(double) * K * N, hipHostMallocDefault);
  hipHostMalloc((void **)&C,sizeof(double) * M * N, hipHostMallocDefault);
  for(i=0;i<M;++i)
    for(j=0;j<K;++j) A[IDX2C(i,j,M)] = i*K+j + 1;
  for(i=0;i<K;++i)
    for(j=0;j<N;++j) B[IDX2C(i,j,K)] = i*N+j + 1;
  for(i=0;i<M;++i)
    for(j=0;j<N;++j) C[IDX2C(i,j,M)] = 0.0;
  cublasAlloc(M*K,sizeof(double),(void **)&dA);
  cublasAlloc(K*N,sizeof(double),(void **)&dB);
  cublasAlloc(M*N,sizeof(double),(void **)&dC);
  hipblasSetMatrix(M,K,sizeof(double),A,LDA,dA,M);
  hipblasSetMatrix(K,N,sizeof(double),B,LDB,dB,K);
  hipblasSetMatrix(M,N,sizeof(double),C,LDC,dC,M);

  cublasDgemm('N','N',M,N,K,alpha,dA,LDA,dB,LDB,beta,dC,LDC);
  hipblasGetMatrix(M,N,sizeof(double),dC,M,C,LDC);
  cublasFree(dA);
  cublasFree(dB);
  cublasFree(dC);
  cublasShutdown();
  return 0;
}
