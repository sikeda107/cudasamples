#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
#include<memory.h>
#include<math.h>
#include"timer.h"
//CUDA library
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void kernel(int size,int* d_vec){
  int thid = threadIdx.x;
  int blid = blockIdx.x;

  /* if(thid < size && blid < 1){
    printf("blid:%d,thid:%d,vec[%d]:%d\n",blid,thid,thid,d_vec[thid]);
    }*/
  if(blid<2&&thid<2)
  printf("blid:%d,thid:%d\n",blid,thid,thid);
}

int main(){

  int* vec;
  //int vec[10];
  int size = 10;
  int* d_vec;
  int i;

  dim3 grid(32);
  dim3 block(32);
 
  //----set up device START-----
  hipDeviceProp_t deviceProp;
  int dev_num =0;
  hipGetDeviceProperties(&deviceProp,dev_num);
  printf("Using Device %d:%s\n",dev_num,deviceProp.name);
  hipSetDevice(dev_num);
  //----set up device START-----

  vec = (int*)malloc(sizeof(int)*size);
 
  //----initialize vec----
  for(i=0;i<size;i++){
    vec[i]=i;
    printf("[%d]:%d\n",i,vec[i]);
  }
  
  hipMalloc((int**)&d_vec,sizeof(int)*size);
  hipMemcpy(d_vec,vec,sizeof(int)*size,hipMemcpyHostToDevice);
  for(i=0;i<size;i++){
    vec[i]=0;
    printf("[%d]:%d\n",i,vec[i]);
  }
 
  kernel<<<grid,block>>>(size,d_vec);
  printf("hoge\n\n");
  kernel<<<grid,block>>>(size,d_vec);

  hipDeviceSynchronize();

  hipMemcpy(vec,d_vec,sizeof(int)*size,hipMemcpyDeviceToHost);
  for(i=0;i<size;i++){
    printf("[%d]:%d\n",i,vec[i]);
  }
  hipFree(d_vec);
  hipDeviceReset();

  free(vec);
  
  return 0;
}

