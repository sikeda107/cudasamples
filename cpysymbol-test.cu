#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
#include<memory.h>
#include<math.h>
#include"timer.h"
//CUDA library
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__device__ int d_vec[3][3][3];
//int *d_vec;

__global__ void kernel( ){
		       //int *d_vec){
  int thid = threadIdx.x;
  int blid = blockIdx.x;
  int i,j,k;

  if(blid<1&&thid<1){
    printf("blid:%d,thid:%d\n",blid,thid,thid);
    for(i=0;i<3;i++){
      for(j=0;j<3;j++){
	for(k=0;k<3;k++){
	  d_vec[i][j][k] = (int)fabs((-2)* pow(2.0,2.0));
	  //d_vec[i*3*3+j*3+k] += 3;
	}//#for
      }//#for
    }//#for
  }//#if
}
int main(){

  int vec[3][3][3];
  int i,j,k;
  int size = 27;
  dim3 grid(32);
  dim3 block(32);
 
  //----set up device START-----
  hipDeviceProp_t deviceProp;
  int dev_num =0;
  hipGetDeviceProperties(&deviceProp,dev_num);
  printf("Using Device %d:%s\n",dev_num,deviceProp.name);
  hipSetDevice(dev_num);
  //----set up device START-----

 
  //----initialize vec----
  for(i=0;i<3;i++){
    for(j=0;j<3;j++){
      for(k=0;k<3;k++){
	vec[i][j][k] = 6;
	printf("vec[%d][%d][%d]:%d\n",i,j,k,vec[i][j][k]);
      }}}
  //hipMalloc((int**)&d_vec,sizeof(int)*size);  
  
  //hipMemcpy(d_vec,vec,sizeof(int)*(size),hipMemcpyHostToDevice); 
    hipMemcpyToSymbol(HIP_SYMBOL(d_vec),vec,sizeof(int)*size);//HOST TO DEVICE
  
    //kernel<<<grid,block>>>(d_vec);
  kernel<<<grid,block>>>();
  
  hipDeviceSynchronize();
 
  //hipMemcpy(vec,d_vec,sizeof(int)*(size),hipMemcpyDeviceToHost); 
 
  hipMemcpyFromSymbol(vec,HIP_SYMBOL(d_vec),sizeof(int)*size);  //DEVICE TO HOST
  
  hipFree(d_vec);
  
  for(i=0;i<3;i++){
    for(j=0;j<3;j++){
      for(k=0;k<3;k++){
	printf("vec[%d][%d][%d]:%d\n",i,j,k,vec[i][j][k]);
      }}}
  

  //hipFree(d_vec);
  hipDeviceReset();

  return 0;
}

